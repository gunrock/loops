/**
 * @file thread_mapped.cu
 * @author 
 * @brief SpGEMM example
 * @version 0.1
 * @date 2023
 *
 * @copyright Copyright (c) 2023
 *
 */

#include "helpers.hxx"
#include <loops/algorithms/spgemm/thread_mapped.cuh>
#include <loops/algorithms/spgemm/estimate_nnz.cuh>
#include <loops/algorithms/spgemm/estimate_nnz_test.cuh>
#include <loops/algorithms/spgemm/find_explicit_zeros.cuh>

#include "test_spgemm.cpp"

using namespace loops;

int main(int argc, char** argv) {
  util::timer_t timer;

  using index_t = int;
  using offset_t = int;
  using type_t = float;

  // ... I/O parameters, mtx, etc.
  parameters_t parameters(argc, argv);

  matrix_market_t<index_t, offset_t, type_t> mtx;
  csr_t<index_t, offset_t, type_t> csr(mtx.load(parameters.filename));
  csc_t<index_t, offset_t, type_t> csc(mtx.load(parameters.filename));

  int* h_nnz_C_by_row = new int[csc.cols]();
  int* d_nnz_C_by_row;

  hipMalloc(&d_nnz_C_by_row, csc.cols * sizeof(int));
  hipMemcpy(d_nnz_C_by_row, h_nnz_C_by_row, csc.cols * sizeof(int), hipMemcpyHostToDevice);

  algorithms::spgemm::estimate_nnz_test(csr, csc, d_nnz_C_by_row);
  hipMemcpy(h_nnz_C_by_row, d_nnz_C_by_row, csc.cols * sizeof(int), hipMemcpyDeviceToHost);

  // copyAndSumEstimateNnzToHost(d_nnz_C_by_row, csc.cols);

  int nnz_C_sum = algorithms::spgemm::sumEstimateNnzC(h_nnz_C_by_row, csc.cols);
  std::cout << "Sum of d_nnz_C: " << nnz_C_sum << std::endl;

  coo_t<index_t, type_t> coo(csr.rows, csc.cols, nnz_C_sum);

  // Apply SpGEMM


/* SpGEMM */

  // Output matrix.
  matrix_t<type_t> C(csr.rows, csc.cols);

  // Run the benchmark.
  timer.start();
  // algorithms::spgemm::thread_mapped(csr, csc, C);
  algorithms::spgemm::thread_mapped(csr, csc, coo);
  timer.stop();

  std::cout << "Elapsed (ms):\t" << timer.milliseconds() << std::endl;


  loops::matrix_t<type_t, loops::memory_space_t::host> h_C;
  copyDeviceMtxToHost(C, h_C);
  writeMtxToFile(h_C, csr.rows, csc.cols, "/home/ychenfei/research/libs/loops/examples/spgemm/export_mtx/test.txt");

}